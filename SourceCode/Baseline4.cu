#include <stdio.h>
#include <stdint.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#define DEBUG 0

typedef enum { SORT_BY_HOST, SORT_BY_THRUST, SORT_SEQUENTIALLY_BY_HOST_USING_PARALLEL_ALGORITHM } Implementation;

#define CHECK(call) {                                                          \
    const hipError_t error = call;                                            \
    if (error != hipSuccess) {                                                \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

struct GpuTimer {
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start() {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop() {
        hipEventRecord(stop, 0);
    }

    float Elapsed() {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void sortByHost(const uint32_t *input, int n, uint32_t *output, int numBits) {
    int nBins = 1 << numBits;

    size_t inputMemSize = n * sizeof(uint32_t);
    uint32_t *in = (uint32_t *)malloc(inputMemSize);
    uint32_t *originalIn = in;
    memcpy(in, input, inputMemSize);
    uint32_t *out = output;

    int *hist = (int *)malloc(nBins * sizeof(int));
    int *histScan = (int *)malloc(nBins * sizeof(int));

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += numBits) {
    	// TODO: Compute histogram
        memset(hist, 0, nBins * sizeof(int));
        for (int index = 0; index < n; index++) {
            int bin = (in[index] >> bit) & (nBins - 1);
            hist[bin]++;
        }

    	// TODO: Scan histogram (exclusively)
        histScan[0] = 0;
        for (int bin = 1; bin < nBins; bin++) {
            histScan[bin] = histScan[bin - 1] + hist[bin - 1];
        }

    	// TODO: Scatter elements to correct locations
        for (int index = 0; index < n; index++) {
            int bin = (in[index] >> bit) & (nBins - 1);
            out[histScan[bin]] = in[index];
            histScan[bin]++;
        }
    	
    	// Swap in and out.
        uint32_t *temp = in;
        in = out;
        out = temp;
    }
    memcpy(output, in, n * sizeof(uint32_t)); 

    free(hist);
    free(histScan);
    free(originalIn);    
}

void sortByThrust(const uint32_t *in, int n, uint32_t *out) {
    thrust::device_vector<uint32_t> dv_out(in, in + n);
	thrust::sort(dv_out.begin(), dv_out.end());
	thrust::copy(dv_out.begin(), dv_out.end(), out);
}

void sortByHostUsingParallelAlgorithm(const uint32_t *input, int n, uint32_t *output, int numBits, int blockSize) {
    int numBlocks = (n - 1) / blockSize + 1;
    int numBins = 1 << numBits;
    
    size_t inputMemSize = n * sizeof(uint32_t);
    uint32_t *source      = (uint32_t *)malloc(inputMemSize);
    uint32_t *destination = (uint32_t *)malloc(inputMemSize);
    memcpy(source, input, inputMemSize);

    size_t histogramBlocksMemSize = (numBlocks * numBins) * sizeof(int);
    int *blockHistograms     = (int *)malloc(histogramBlocksMemSize);
    int *blockHistogramsScan = (int *)malloc(histogramBlocksMemSize);

    int      *block1BitScan = (int *)     malloc(blockSize * sizeof(int));
    uint32_t *sortedBlock   = (uint32_t *)malloc(blockSize * sizeof(uint32_t));
    int      *firstIndices  = (int *)     malloc(numBins   * sizeof(int));

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += numBits) {
        memset(blockHistograms, 0, histogramBlocksMemSize);

        // Calculate local histogram for each block.
        for (int blockIndex = 0; blockIndex < numBlocks; ++blockIndex) {
            uint32_t *blockPtr = source + (blockIndex * blockSize);
            int *blockHistogram = blockHistograms + (blockIndex * numBins);
            for (int localIndex = 0; localIndex < blockSize; ++localIndex) {
                int index = localIndex + blockIndex * blockSize;
                if (index < n) {
                    int bin = (blockPtr[localIndex] >> bit) & (numBins - 1);
                    blockHistogram[bin]++;
                }
            }
        }

        // Scan histograms by column-major order.
        blockHistogramsScan[0] = 0;
        for (int index = 1; index < numBins * numBlocks; ++index) {
            int currentBin = index / numBlocks;
            int currentBlockIndex = index % numBlocks;

            int previousBin = (index - 1) / numBlocks;
            int previousBlockIndex = (index - 1) % numBlocks;

            blockHistogramsScan[currentBlockIndex * numBins + currentBin] = 
                blockHistogramsScan[previousBlockIndex * numBins + previousBin]
                +   blockHistograms[previousBlockIndex * numBins + previousBin];
        }

        // Sort internally each block, then calculate rank and scatter.
        for (int blockIndex = 0; blockIndex < numBlocks; ++blockIndex) {
            uint32_t *blockPtr = source + (blockIndex * blockSize);

            for (int innerBit = 0; innerBit < numBits; ++innerBit) {
                block1BitScan[0] = 0;
                for (int localIndex = 1; localIndex < blockSize; ++localIndex) {
                    int index = localIndex + blockIndex * blockSize;
                    int inputValue = 0;
                    if (index < n) {
                        inputValue = (blockPtr[localIndex - 1] >> (bit + innerBit)) & 0b1;
                    }
                    block1BitScan[localIndex] = 
                        block1BitScan[localIndex - 1] + inputValue;
                }

                int numZeros;
                if (blockIndex != numBlocks - 1) {
                    numZeros = blockSize
                        - block1BitScan[blockSize - 1]
                        - ((blockPtr[blockSize - 1] >> (bit + innerBit)) & 0b1);
                } else {
                    int realBlockSize = n - (blockIndex * blockSize);
                    numZeros = realBlockSize
                        - block1BitScan[realBlockSize - 1]
                        - ((blockPtr[realBlockSize - 1] >> (bit + innerBit)) & 0b1);
                }
                for (int localIndex = 0; localIndex < blockSize; ++localIndex) {
                    int index = localIndex + blockIndex * blockSize;
                    if (index < n) {
                        int bitValue = (blockPtr[localIndex] >> (bit + innerBit)) & 0b1;
                        int rank;
                        if (bitValue == 0) {
                            rank = localIndex - block1BitScan[localIndex];
                        } else {
                            rank = numZeros + block1BitScan[localIndex];
                        }
                        sortedBlock[rank] = blockPtr[localIndex];
                    }
                }
                for (int localIndex = 0; localIndex < blockSize; ++localIndex) {
                    int index = localIndex + blockIndex * blockSize;
                    if (index < n) {
                        blockPtr[localIndex] = sortedBlock[localIndex];
                    }
                }
            }

            firstIndices[(blockPtr[0] >> bit) & (numBins - 1)] = 0;
            for (int localIndex = 1; localIndex < blockSize; ++localIndex) {
                int index = localIndex + blockIndex * blockSize;
                if (index < n) {
                    int currentValue  = (blockPtr[localIndex]     >> bit) & (numBins - 1);
                    int previousValue = (blockPtr[localIndex - 1] >> bit) & (numBins - 1);
                    if (currentValue != previousValue) {
                        firstIndices[currentValue] = localIndex;
                    }
                }
            }

            for (int localIndex = 0; localIndex < blockSize; ++localIndex) {
                int index = localIndex + blockIndex * blockSize;
                if (index < n) {
                    int currentValue = (blockPtr[localIndex] >> bit) & (numBins - 1);
                    int rank = blockHistogramsScan[blockIndex * numBins + currentValue]
                               + localIndex - firstIndices[currentValue];
                    destination[rank] = blockPtr[localIndex];
                }
            }
        }

        uint32_t *temp = source;
        source = destination;
        destination = temp;
    }
    memcpy(output, source, inputMemSize);

    free(firstIndices);
    free(sortedBlock);
    free(block1BitScan);
    free(blockHistogramsScan);
    free(blockHistograms);
    free(destination);
    free(source);
}

void sort(const uint32_t *in, int n,
          uint32_t *out,
          Implementation implementation = SORT_BY_HOST,
          int numBits = 4,
          int blockSize = 1) {
    GpuTimer timer; 
    timer.Start();

    if (implementation == SORT_BY_HOST) {
    	printf("\nRadix Sort by host\n");
        sortByHost(in, n, out, numBits);
    } else if (implementation == SORT_BY_THRUST) {
    	printf("\nRadix Sort by Thrust library\n");
        sortByThrust(in, n, out);
    } else {
        printf("\nRadix Sort by host, using parallel algorithm\n");
        sortByHostUsingParallelAlgorithm(in, n, out, numBits, blockSize);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo() {
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n) {
    for (int i = 0; i < n; i++) {
        if (out[i] != correctOut[i]) {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%i ", a[i]);
    }
    printf("\n");
}

int main(int argc, char ** argv) {
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n;
    if (DEBUG) {
        n = 513;
    } else {
        n = (1 << 24) + 1;
    }
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t *in = (uint32_t *)malloc(bytes);
    uint32_t *out = (uint32_t *)malloc(bytes);
    uint32_t *correctOut = (uint32_t *)malloc(bytes);

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++) {
        if (DEBUG) {
            in[i] = rand() & 0xFF;
        } else {
            in[i] = rand();
        }
    }
    if (DEBUG) {
        printArray(in, n);
    }

    // DETERMINE BLOCK SIZE
    int blockSize = 512;
    if (argc == 2) {
        blockSize = atoi(argv[1]);
    }

    int numBits = 4;
    if (argc == 3) {
        numBits = atoi(argv[2]);
    }

    // Sorting by Host
    sort(in, n, correctOut, SORT_BY_HOST, numBits);
    if (DEBUG) {
        printArray(correctOut, n);
    }

    // Sorting by Thrust Library
    sort(in, n, out, SORT_BY_THRUST);
    if (DEBUG) {
        printArray(out, n);
    }
    checkCorrectness(out, correctOut, n);
    memset(out, 0u, n * sizeof(uint32_t)); // Reset ouput.

    // Sorting by Device
    sort(in, n, out, SORT_SEQUENTIALLY_BY_HOST_USING_PARALLEL_ALGORITHM, numBits, blockSize);
    if (DEBUG) {
        printArray(out, n);
    }
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
