#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include "common/common.h"

#define DEVICE_ID 0
#define DEBUG 0
#define ENABLE_CHECK_ERROR 1
#define MEASURE_PORTION_EXECUTION_TIME 1

#if !ENABLE_CHECK_ERROR
    #undef CHECK // Remove CHECK macro.
    #define CHECK
#endif

#define TRANSPOSE_SMEM_PAD 2

typedef enum { SORT_BY_HOST, SORT_BY_THRUST, SORT_BY_DEVICE } Implementation;

void sortByHost(const uint32_t *input, int n, uint32_t *output, int numBits) {
    int nBins = 1 << numBits;

    size_t inputMemSize = n * sizeof(uint32_t);
    uint32_t *in = (uint32_t *)malloc(inputMemSize);
    uint32_t *originalIn = in;
    memcpy(in, input, inputMemSize);
    uint32_t *out = output;

    int *hist = (int *)malloc(nBins * sizeof(int));
    int *histScan = (int *)malloc(nBins * sizeof(int));

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += numBits) {
    	// TODO: Compute histogram
        memset(hist, 0, nBins * sizeof(int));
        for (int index = 0; index < n; index++) {
            int bin = (in[index] >> bit) & (nBins - 1);
            hist[bin]++;
        }

    	// TODO: Scan histogram (exclusively)
        histScan[0] = 0;
        for (int bin = 1; bin < nBins; bin++) {
            histScan[bin] = histScan[bin - 1] + hist[bin - 1];
        }

    	// TODO: Scatter elements to correct locations
        for (int index = 0; index < n; index++) {
            int bin = (in[index] >> bit) & (nBins - 1);
            out[histScan[bin]] = in[index];
            histScan[bin]++;
        }

    	// Swap in and out.
        uint32_t *temp = in;
        in = out;
        out = temp;
    }
    memcpy(output, in, n * sizeof(uint32_t));

    free(hist);
    free(histScan);
    free(originalIn);
}

void sortByThrust(const uint32_t *input, int n, uint32_t *output) {
    thrust::device_vector<uint32_t> d_output(input, input + n);
	thrust::sort(d_output.begin(), d_output.end());
	thrust::copy(d_output.begin(), d_output.end(), output);
}

__device__ __forceinline__ uint32_t mask(uint32_t number, int startBit, int numBits) {
    return (number >> startBit) & ((0b1 << numBits) - 1);
}

__global__ void scanLocallyBlocksUnroll2Kernel(const uint32_t *g_input, int n, uint32_t *g_output, int bit) {
    extern __shared__ uint32_t s_data[];
    
    int index = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // 1. Load data handled by this block into SMEM.
    int indexPart1 = index;
    int indexPart2 = blockDim.x + index;
    int sIndexPart1 = threadIdx.x;
    int sIndexPart2 = blockDim.x + threadIdx.x;
    if (indexPart1 < n) {
        s_data[sIndexPart1] = mask(g_input[indexPart1], bit, 1);
    } else {
        s_data[sIndexPart1] = 0;
    }

    if (indexPart2 < n) {
        s_data[sIndexPart2] = mask(g_input[indexPart2], bit, 1);
    } else {
        s_data[sIndexPart2] = 0;
    }

    // 2. Do scan with data on SMEM.
    // >>>> Up-Sweep phase.
    int offset = 1;
    for (int nNodes = blockDim.x; nNodes > 0; nNodes >>= 1) {
        __syncthreads();
        if (threadIdx.x < nNodes) {
            int sIndexRight = threadIdx.x * 2 * offset + offset * 2 - 1;
            int sIndexLeft  = threadIdx.x * 2 * offset + offset - 1;
            s_data[sIndexRight] += s_data[sIndexLeft];
        }
        offset *= 2;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        s_data[blockDim.x * 2 - 1] = 0;
    }

    // >>>> Down-Sweep phase.
    for (int nNodes = 1; nNodes <= blockDim.x; nNodes *= 2) {
        __syncthreads();
        offset >>= 1;
        if (threadIdx.x < nNodes) {
            int sIndexRight = threadIdx.x * 2 * offset + offset * 2 - 1;
            int sIndexLeft  = threadIdx.x * 2 * offset + offset - 1;
            uint32_t temp = s_data[sIndexRight];
            s_data[sIndexRight] += s_data[sIndexLeft];
            s_data[sIndexLeft] = temp;
        }
    }
    __syncthreads();

    // 3. Copy back results from SMEM to GMEM.
    if (indexPart1 < n) {
        g_output[indexPart1] = s_data[sIndexPart1];
    }

    if (indexPart2 < n) {
        g_output[indexPart2] = s_data[sIndexPart2];
    }
}

__global__ void scatterLocallyBlocksKernel(const uint32_t *g_input, int n, const uint32_t *g_scan, uint32_t *g_output, int bit) {
    extern __shared__ uint32_t s_output[];

    int index = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    const uint32_t *g_blockInput = g_input + blockIdx.x * blockDim.x * 2;
    const uint32_t *g_blockScan = g_scan + blockIdx.x * blockDim.x * 2;
    uint32_t *g_blockOutput = g_output + blockIdx.x * blockDim.x * 2;

    int blockSize;
    if (blockIdx.x != gridDim.x - 1) {
        blockSize = blockDim.x * 2;
    } else {
        blockSize = n - (blockIdx.x * blockDim.x * 2);
    }
    int numZeros = blockSize - g_blockScan[blockSize - 1] - mask(g_blockInput[blockSize - 1], bit, 1);

    if (index < n) {
        uint32_t element = g_blockInput[threadIdx.x];
        int rank;
        if (mask(element, bit, 1) == 0) {
            rank = threadIdx.x - g_blockScan[threadIdx.x];
        } else {
            rank = numZeros + g_blockScan[threadIdx.x];
        }

        s_output[rank] = element;
    }

    if (index + blockDim.x < n) {
        int element = g_blockInput[blockDim.x + threadIdx.x];
        int rank;
        if (mask(element, bit, 1) == 0) {
            rank = blockDim.x + threadIdx.x - g_blockScan[blockDim.x + threadIdx.x];
        } else {
            rank = numZeros + g_blockScan[blockDim.x + threadIdx.x];
        }

        s_output[rank] = element;    
    }
    __syncthreads();

    if (index < n) {
        g_blockOutput[threadIdx.x] = s_output[threadIdx.x];
    }

    if (blockDim.x + index < n) {
        g_blockOutput[blockDim.x + threadIdx.x] = s_output[blockDim.x + threadIdx.x];
    }
}

void sortLocallyDataBlocks(const uint32_t *d_input, int n, uint32_t *d_output, int startBit, int numBits, int blockSize) {
    // Scan kernel implementation with unroll 2.
    dim3 scanBlockDim(blockSize);
    dim3 scanGridDim((n - 1) / (scanBlockDim.x * 2) + 1);
    size_t scanSharedMemory = scanBlockDim.x * 2 * sizeof(uint32_t);    

    dim3 scatterBlockDim(blockSize);
    dim3 scatterGridDim((n - 1) / (scatterBlockDim.x * 2) + 1);
    size_t scatterSharedMemory = scatterBlockDim.x * 2 * sizeof(uint32_t);

    static uint32_t *d_inputMirror = NULL;
    size_t inputMemSize = n * sizeof(uint32_t);
    if (d_inputMirror == NULL) {
        CHECK(hipMalloc(&d_inputMirror, inputMemSize));
    }
    hipMemcpy(d_inputMirror, d_input, inputMemSize, hipMemcpyDeviceToDevice);

    static uint32_t *d_outputMirror = NULL;
    if (d_outputMirror == NULL) {
        CHECK(hipMalloc(&d_outputMirror, inputMemSize));
    }

    static uint32_t *d_scan = NULL;
    if (d_scan == NULL) {
        CHECK(hipMalloc(&d_scan, inputMemSize));
    }

    for (int bit = startBit; bit < startBit + numBits; ++bit) {
        scanLocallyBlocksUnroll2Kernel<<<
            scanGridDim, scanBlockDim, scanSharedMemory
        >>>(d_inputMirror, n, d_scan, bit);
        #if ENABLE_CHECK_ERROR
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        #endif
        
        scatterLocallyBlocksKernel<<<
            scatterGridDim, scatterBlockDim, scatterSharedMemory
        >>>(d_inputMirror, n, d_scan, d_outputMirror, bit);
        #if ENABLE_CHECK_ERROR
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        #endif

        uint32_t *temp = d_inputMirror;
        d_inputMirror = d_outputMirror;
        d_outputMirror = temp;
    }
    CHECK(hipMemcpy(d_output, d_inputMirror, inputMemSize, hipMemcpyDeviceToDevice));

    if (startBit + numBits == sizeof(uint32_t) - 1) {
        hipFree(d_inputMirror);
        hipFree(d_outputMirror);
        hipFree(d_scan);
        d_inputMirror = NULL;
        d_outputMirror = NULL;
        d_scan = NULL;
    }
}

__global__ void scatterKernel(const uint32_t *g_input, int n, uint32_t *g_output, uint32_t *g_histogramTableScan, int startBit, int numBits) {
    extern __shared__ uint8_t sharedMemory[];
    uint32_t *s_input = (uint32_t *)sharedMemory;
    uint32_t *s_firstIndices = (uint32_t *)(sharedMemory + blockDim.x * 2 * sizeof(uint32_t));

    int index = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    if (index < n) {
        s_input[threadIdx.x] = g_input[index];
    }
    if (index + blockDim.x < n) {
        s_input[blockDim.x + threadIdx.x] = g_input[index + blockDim.x];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        s_firstIndices[
            mask(s_input[0], startBit, numBits)
        ] = 0;
    }

    if (threadIdx.x > 0 && index < n) {
        uint32_t currentValue  = mask(s_input[threadIdx.x], startBit, numBits);
        uint32_t previousValue = mask(s_input[threadIdx.x - 1], startBit, numBits);
        if (currentValue != previousValue) {
            s_firstIndices[currentValue] = threadIdx.x;
        }
    }

    if (index + blockDim.x < n) {
        uint32_t currentValue  = mask(s_input[blockDim.x + threadIdx.x], startBit, numBits);
        uint32_t previousValue = mask(s_input[blockDim.x + threadIdx.x - 1], startBit, numBits);
        if (currentValue != previousValue) {
            s_firstIndices[currentValue] = blockDim.x + threadIdx.x;
        }
    }
    __syncthreads();

    int numBins = 1 << numBits;
    if (index < n) {
        uint32_t currentValue  = mask(s_input[threadIdx.x], startBit, numBits);
        uint32_t rank = g_histogramTableScan[blockIdx.x * numBins + currentValue]
                            + threadIdx.x - s_firstIndices[currentValue];
        g_output[rank] = s_input[threadIdx.x];
    }

    if (index + blockDim.x < n) {
        uint32_t currentValue  = mask(s_input[blockDim.x + threadIdx.x], startBit, numBits);
        uint32_t rank = g_histogramTableScan[blockIdx.x * numBins + currentValue]
                            + (blockDim.x + threadIdx.x) - s_firstIndices[currentValue];
        g_output[rank] = s_input[blockDim.x + threadIdx.x];
    }
}

void scatter(const uint32_t *d_input, int n, uint32_t *d_output, uint32_t *d_histogramTableScan, int startBit, int numBits, int blockSize) {   
    dim3 blockDim(blockSize);
    dim3 gridDim((n - 1) / (blockDim.x * 2) + 1);
    int numBins = 1 << numBits;
    size_t sharedMemSize = blockDim.x * 2 * sizeof(uint32_t) + numBins * sizeof(uint32_t);    
    scatterKernel<<<gridDim, blockDim, sharedMemSize>>>(d_input, n, d_output, d_histogramTableScan, startBit, numBits);
    #if ENABLE_CHECK_ERROR
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    #endif
}

__global__ void histogramKernel(const uint32_t *g_input, int n, uint32_t *g_histogramTable, int startBit, int numBits) {
    extern __shared__ uint32_t s_localHistogram[];
    
    size_t numBins = 0b1 << numBits;
    for (int bin = threadIdx.x; bin < numBins; bin += blockDim.x) {
        s_localHistogram[bin] = 0;
    }
    __syncthreads();

    int index = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    if (index < n) {
        int bin = mask(g_input[index], startBit, numBits);
        atomicAdd(&s_localHistogram[bin], 1);
    }

    if (index + blockDim.x < n) {
        int bin = mask(g_input[index + blockDim.x], startBit, numBits);
        atomicAdd(&s_localHistogram[bin], 1);
    }
    __syncthreads();
    
    uint32_t *localHistogram = g_histogramTable + blockIdx.x * numBins;
    for (int bin = threadIdx.x; bin < numBins; bin += blockDim.x) {
        atomicAdd(&localHistogram[bin], s_localHistogram[bin]);
    }
}

void histogram(const uint32_t *d_input, int n, uint32_t *d_histogramTable, int startBit, int numBits, int blockSize) {
    dim3 blockDim(blockSize);
    dim3 gridDim((n - 1) / (blockDim.x * 2) + 1);
    
    size_t numBins = 0b1 << numBits;
    size_t histogramTableMemSize = gridDim.x * numBins * sizeof(uint32_t);
    CHECK(hipMemset(d_histogramTable, 0, histogramTableMemSize));

    size_t sharedMemSize = numBins * sizeof(uint32_t);
    histogramKernel<<<gridDim, blockDim, sharedMemSize>>>(d_input, n, d_histogramTable, startBit, numBits);
    #if ENABLE_CHECK_ERROR
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    #endif
}

__global__ void transposedKernel(const uint32_t *g_input, int numCols, int numRows, uint32_t *g_output) {
    extern __shared__ uint32_t s_tile[];

    int inCol = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    int inRow = blockIdx.y * blockDim.y + threadIdx.y;
    int inIndex = inRow * numCols + inCol;
    
    int insideBlockIndex = threadIdx.y * (blockDim.x * 2 + TRANSPOSE_SMEM_PAD) + threadIdx.x;
    int insideBlockIndexWithoutPad = threadIdx.y * blockDim.x + threadIdx.x;
    int insideTransposedBlockCol = insideBlockIndexWithoutPad % blockDim.y;
    int insideTransposedBlockRow = insideBlockIndexWithoutPad / blockDim.y;
    int insideTransposedBlockIndex = insideTransposedBlockCol * (blockDim.x * 2 + TRANSPOSE_SMEM_PAD) + insideTransposedBlockRow;

    int outCol = blockIdx.y * blockDim.y + insideTransposedBlockCol;
    int outRow = blockIdx.x * (blockDim.x * 2) + insideTransposedBlockRow;
    int outIndex = outRow * numRows + outCol;

    if (inCol < numCols && inRow < numRows) {
        s_tile[insideBlockIndex] = g_input[inIndex];
    }
    if (inCol + blockDim.x < numCols && inRow < numRows) {
        s_tile[insideBlockIndex + blockDim.x] = g_input[inIndex + blockDim.x];
    }
    __syncthreads();

    if (outCol < numRows && outRow < numCols) {
        g_output[outIndex] = s_tile[insideTransposedBlockIndex];
    }
    if (outCol < numRows && outRow + blockDim.x < numCols) {
        g_output[outIndex + numRows * blockDim.x] = s_tile[insideTransposedBlockIndex + blockDim.x];
    }
}

void transpose(const uint32_t *d_input, int numCols, int numRows, uint32_t *d_output) {
    dim3 blockDim(32, 32);
    dim3 gridDim(
        (numCols - 1) / (blockDim.x * 2) + 1,
        (numRows - 1) / blockDim.y + 1
    );
    size_t sharedMemSize = blockDim.y * (blockDim.x * 2 + TRANSPOSE_SMEM_PAD) * sizeof(uint32_t);
    transposedKernel<<<gridDim, blockDim, sharedMemSize>>>(d_input, numCols, numRows, d_output);
    #if ENABLE_CHECK_ERROR
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    #endif
}

__global__ void scanBlocks(const uint32_t *g_input, int n, uint32_t *g_output, uint32_t *blockSums) {
    // SMEM Size: blockDim.x elements.
    extern __shared__ uint32_t s_data[];
    
    int index = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // 1. Load data handled by this block into SMEM.
    if (index < n) {
        s_data[threadIdx.x] = g_input[index];
    } else {
        s_data[threadIdx.x] = 0;
    }

    if (index + blockDim.x < n) {
        s_data[blockDim.x + threadIdx.x] = g_input[index + blockDim.x];
    } else {
        s_data[blockDim.x + threadIdx.x] = 0;
    }

    // 2. Do scan with data on SMEM. Implementation of Work-Efficient algorithm.
    // >>>> Up-Sweep phase.
    int offset = 1; // Distance between 2 adjacent nodes will be added together, in the current level.
    for (int nNodes = blockDim.x; nNodes > 0; nNodes >>= 1) { // nNodes in the number of parent nodes, in the upper level.
        __syncthreads();
        if (threadIdx.x < nNodes) {
            int sIndexRight = threadIdx.x * 2 * offset + offset * 2 - 1;
            int sIndexLeft  = threadIdx.x * 2 * offset + offset - 1;
            s_data[sIndexRight] += s_data[sIndexLeft];
        }
        offset *= 2;
    }
    __syncthreads();

    if (threadIdx.x == 0 && blockSums != NULL) {
        // Copy sum of block into block sums array.
        blockSums[blockIdx.x] = s_data[blockDim.x * 2 - 1];
    }
    if (threadIdx.x == 0) {
        // Set 0 for the last element.
        s_data[blockDim.x * 2 - 1] = 0;
    }

    // >>>> Down-Sweep phase.
    for (int nNodes = 1; nNodes <= blockDim.x; nNodes *= 2) {
        __syncthreads();
        offset >>= 1;
        if (threadIdx.x < nNodes) {
            int sIndexRight = threadIdx.x * 2 * offset + offset * 2 - 1;
            int sIndexLeft  = threadIdx.x * 2 * offset + offset - 1;
            uint32_t temp = s_data[sIndexRight];
            s_data[sIndexRight] += s_data[sIndexLeft];
            s_data[sIndexLeft] = temp;
        }
    }
    __syncthreads();

    // 3. Copy back results from SMEM to GMEM.
    if (index < n) {
        g_output[index] = s_data[threadIdx.x];
    }

    if (index + blockDim.x < n) {
        g_output[index + blockDim.x] = s_data[blockDim.x + threadIdx.x];
    }
}

__global__ void addScannedBlockSumsToScannedBlocks(uint32_t *blockSums, uint32_t *blockScans, int n) {
    int index = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    index += blockDim.x * 2; // Shift to right 1 block.
    if (index < n) {
        blockScans[index] += blockSums[blockIdx.x];
    }
    if (index + blockDim.x < n) {
        blockScans[index + blockDim.x] += blockSums[blockIdx.x];
    }
}

void scan(const uint32_t *d_input, int n, uint32_t *d_output, int blockSize) {
    dim3 blockDim(blockSize);
    dim3 gridDim((n - 1) / (blockDim.x * 2) + 1);
    
    static uint32_t *d_blockSums = NULL;
    static uint32_t *h_blockSums = NULL;
    static size_t blockSumsMemSize = 0;

    size_t newBlockSumsMemSize = gridDim.x * sizeof(uint32_t);
    if (gridDim.x > 1 && blockSumsMemSize != newBlockSumsMemSize)  {
        if (d_blockSums != NULL) {
            CHECK(hipFree(d_blockSums));
        }
        if (h_blockSums != NULL) {
            delete[] h_blockSums;
        }

        blockSumsMemSize = newBlockSumsMemSize;
        h_blockSums = new uint32_t[gridDim.x];
        CHECK(hipMalloc(&d_blockSums, blockSumsMemSize));
    }

    size_t sharedMemSize = blockDim.x * 2 * sizeof(uint32_t);
    scanBlocks<<<gridDim, blockDim, sharedMemSize>>>(d_input, n, d_output, d_blockSums);
    #if ENABLE_CHECK_ERROR
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    #endif

    if (gridDim.x > 1) {
        hipMemcpy(h_blockSums, d_blockSums, blockSumsMemSize, hipMemcpyDeviceToHost);
        for (int index = 1; index < gridDim.x; ++index) {
            h_blockSums[index] += h_blockSums[index - 1];
        }
        hipMemcpy(d_blockSums, h_blockSums, blockSumsMemSize, hipMemcpyHostToDevice);

        dim3 postScanBlocksGridDim(gridDim.x - 1);
        addScannedBlockSumsToScannedBlocks<<<postScanBlocksGridDim, blockDim>>>(d_blockSums, d_output, n);
        #if ENABLE_CHECK_ERROR
        CHECK(hipDeviceSynchronize());
        CHECK(hipGetLastError());
        #endif
    }
}

void sortByDevice(const uint32_t *h_input, int n, uint32_t *h_output, int numBits, int blockSize) {
    // ============================================
    #if MEASURE_PORTION_EXECUTION_TIME
    GpuTimer timer;
    float sortingLocallyBlocksElapsedTime = 0.0f;
    float histogramElapsedTime = 0.0f;
    float scanElapsedTime = 0.0f;
    float scatteringElapsedTime = 0.0f;
    #endif
    // ============================================

    int numBlocks = (n - 1) / (blockSize * 2) + 1;
    int numBins = 1 << numBits;
    
    uint32_t *d_input  = NULL;
    uint32_t *d_output = NULL;
    size_t inputMemSize = n * sizeof(uint32_t);
    CHECK(hipMalloc(&d_input, inputMemSize));
    CHECK(hipMalloc(&d_output, inputMemSize));
    CHECK(hipMemcpy(d_input, h_input, inputMemSize, hipMemcpyHostToDevice));

    uint32_t *d_histogramTable              = NULL;
    uint32_t *d_histogramTableTranspose     = NULL;
    uint32_t *d_histogramTableScanTranspose = NULL;
    uint32_t *d_histogramTableScan          = NULL;
    size_t histogramTableMemSize = (numBlocks * numBins) * sizeof(uint32_t);
    CHECK(hipMalloc(&d_histogramTable, histogramTableMemSize));
    CHECK(hipMalloc(&d_histogramTableTranspose, histogramTableMemSize));
    CHECK(hipMalloc(&d_histogramTableScanTranspose, histogramTableMemSize));
    CHECK(hipMalloc(&d_histogramTableScan, histogramTableMemSize));

    for (int startBit = 0; startBit < sizeof(uint32_t) * 8; startBit += numBits) {
        // ============================================
        // Sort internally each block.
        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Start();
        #endif
        
        sortLocallyDataBlocks(d_input, n, d_input, startBit, numBits, blockSize);

        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Stop();
        sortingLocallyBlocksElapsedTime += timer.Elapsed();
        #endif
        // ============================================

        // ============================================
        // Calculate local histogram for each block.
        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Start();
        #endif

        histogram(d_input, n, d_histogramTable, startBit, numBits, blockSize);

        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Stop();
        histogramElapsedTime += timer.Elapsed();
        #endif
        // ============================================

        // ============================================
        // Scan histograms by column-major order.
        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Start();
        #endif

        transpose(d_histogramTable, numBins, numBlocks, d_histogramTableTranspose);
        scan(d_histogramTableTranspose, numBlocks * numBins, d_histogramTableScanTranspose, blockSize);
        transpose(d_histogramTableScanTranspose, numBlocks, numBins, d_histogramTableScan);
        
        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Stop();
        scanElapsedTime += timer.Elapsed();
        #endif
        // ============================================

        // ============================================
        // Calculate rank and scatter.
        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Start();
        #endif

        scatter(d_input, n, d_output, d_histogramTableScan, startBit, numBits, blockSize);

        #if MEASURE_PORTION_EXECUTION_TIME
        timer.Stop();
        scatteringElapsedTime += timer.Elapsed();
        #endif
        // ============================================

        uint32_t *temp = d_input;
        d_input = d_output;
        d_output = temp;
    }
    CHECK(hipMemcpy(h_output, d_input, inputMemSize, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_histogramTableScan));
    CHECK(hipFree(d_histogramTableScanTranspose));
    CHECK(hipFree(d_histogramTableTranspose));
    CHECK(hipFree(d_histogramTable));
    CHECK(hipFree(d_output));
    CHECK(hipFree(d_input));

    #if MEASURE_PORTION_EXECUTION_TIME
    printf(">>>> Time | Sort locally blocks   : %.3f\n", sortingLocallyBlocksElapsedTime);
    printf(">>>> Time | Histogram             : %.3f\n", histogramElapsedTime);
    printf(">>>> Time | Scan                  : %.3f\n", scanElapsedTime);
    printf(">>>> Time | Scatter               : %.3f\n", scatteringElapsedTime);
    #endif
}

void sort(const uint32_t *in, int n,
          uint32_t *out,
          Implementation implementation = SORT_BY_HOST,
          int numBits = 4,
          int blockSize = 1) {
    GpuTimer timer;
    timer.Start();

    if (implementation == SORT_BY_HOST) {
    	printf("\nRadix Sort by host\n");
        sortByHost(in, n, out, numBits);
    } else if (implementation == SORT_BY_THRUST) {
    	printf("\nRadix Sort by Thrust library\n");
        sortByThrust(in, n, out);
    } else {
        printf("\nRadix Sort by device:\n");
        sortByDevice(in, n, out, numBits, blockSize);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo() {
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t *out, uint32_t *correctOut, int n) {
    for (int i = 0; i < n; i++) {
        if (out[i] != correctOut[i]) {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t *a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%i ", a[i]);
    }
    printf("\n");
}

int main(int argc, char **argv) {
    CHECK(hipSetDevice(DEVICE_ID));
    CHECK(hipDeviceReset());
    hipFree(0);
    
    printDeviceInfo();

    // Initialization of input.
    int n;
    #if DEBUG
    n = 513;
    #else
    n = (1 << 24) + 1;
    #endif
    printf("\nInput size: %d\n", n);

    size_t inputMemSize = n * sizeof(uint32_t);
    uint32_t *input = (uint32_t *)malloc(inputMemSize);
    uint32_t *output = (uint32_t *)malloc(inputMemSize);
    uint32_t *correctOutput = (uint32_t *)malloc(inputMemSize);

    for (int i = 0; i < n; i++) {
        #if DEBUG
        input[i] = rand() & 0xFF;
        #else
        input[i] = rand();
        #endif
    }
    #if DEBUG
    printArray(input, n);
    #endif

    // Block size.
    int blockSize = 512;
    if (argc > 1) {
        blockSize = atoi(argv[1]);
    }
    printf("Block size: %d\n", blockSize);

    // Digit width.
    int numBits;
    #if DEBUG
    numBits = 4;
    #else
    numBits = 8;
    #endif
    if (argc > 2) {
        numBits = atoi(argv[2]);
    }
    printf("Digit width: %d-bit\n", numBits);

    // Sorting by Host
    sort(input, n, correctOutput, SORT_BY_HOST, numBits);
    #if DEBUG
    printArray(correctOutput, n);
    #endif

    // Sorting by Thrust Library.
    memset(output, 0u, inputMemSize);
    sort(input, n, output, SORT_BY_THRUST);
    #if DEBUG
    printArray(output, n);
    #endif
    checkCorrectness(output, correctOutput, n);

    // Sorting by Device.
    memset(output, 0u, inputMemSize);
    sort(input, n, output, SORT_BY_DEVICE, numBits, blockSize);
    #if DEBUG
    printArray(output, n);
    #endif
    checkCorrectness(output, correctOutput, n);

    free(input);
    free(output);
    free(correctOutput);

    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
